
#include <hip/hip_runtime.h>
__global__ void sineWave_kernel(float4 *pos, unsigned int mesh_width, unsigned int mesh_height, float animation_time)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / float(mesh_width);
	float v = y / float(mesh_height);
	
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	float frequency = 4.0f;
	float w = sinf(u * frequency + animation_time) * cosf(v * frequency + animation_time) + 0.5f;

	pos[y * mesh_width + x] = make_float4(u, w, v, 1.0f);
}

void launchCUDAKernel(float4 *pos, unsigned int mesh_width, unsigned int mesh_height, float type)
{
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width/block.x, mesh_height/block.y, 1);
	sineWave_kernel <<< grid, block >>> (pos, mesh_width, mesh_height, type);
}